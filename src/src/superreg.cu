#include "hip/hip_runtime.h"
#include "Python.h"
#include <stdlib.h>
#define NPY_NO_DEPRECATED_API NPY_1_9_API_VERSION
#include "arrayobject.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
//#include <hip/hip_runtime_api.h> //In samples/common/inc
//#include <thrust/device_vector.h>
//#include <thrust/functional.h>
//#include <thrust/transform_reduce.h>

#define CUDA_DEVICE 0

void init_cuda();
static PyObject *superreg_chebpoly(PyObject *self, PyObject *args);

const char docstring[] = ""
"Evaluate all terms of a chebyshev interpolant, but not summing the terms\n"
"when done. This is useful for evaluating fit Jacobians. This function only\n"
"applies to 2D images which only involve shifting the same set of coeff.\n"
"\n"
"Parameters\n"
"----------\n"
"coefficients : ndarray\n"
"   Cheb coefficients in the format [Cy, Cx]\n"
"\n"
"coordinates : ndarray\n"
"   Coordinates over which to evaluate in the shape [images, 2, ny, nx]\n"
"\n"
"devicearray : ndarray\n"
"   Array of pointers to device memory, created with `device_arrays`\n"
"\n"
"Returns\n"
"-------\n"
"polyterms : ndarray\n"
"   Each cheb term in the shape [Cy, Cx, images, ny, nx]\n";

// Method definition object for this extension, these argumens mean:
// ml_name: The name of the method
// ml_meth: Function pointer to the method implementation
// ml_flags: Flags indicating special features of this method, such as
//          accepting arguments, accepting keyword arguments, being a
//          class method, or being a static method of a class.
// ml_doc:  Contents of this method's docstring
static PyMethodDef superreg_methods[] = {
    {"chebpoly", superreg_chebpoly, METH_VARARGS, docstring},
    {NULL, NULL, 0, NULL}
};

static struct PyModuleDef superreg_definition = {
    PyModuleDef_HEAD_INIT,
    "superreg_chebpoly",
    "A Python module using CUDA for Super Registration",
    -1,
    superreg_methods
};

PyMODINIT_FUNC PyInit__superreg(void) {
    PyObject *module = PyModule_Create(&superreg_definition);
    import_array();
    init_cuda();
	return module;
}

#define COEFFICIENT_CACHE_SIZE 16

/*
bdimx, bdimy determined by CI*CJ, limited to 512 in each dimension (ideally
multiple of 32)
gdimx, gdimy determined by NX*NY*L / CI*CJ, limited to 2**16 in each dimesion
*/
__global__ void cu_chebpoly(float *coef, float *coordx, float *coordy,
        float *terms, int L, int CI, int CJ, int NX, int NY){
    /* calculate all the block indentifier stuff */
    int tidx = threadIdx.x, tidy = threadIdx.y;
    int bdimx = blockDim.x, bdimy = blockDim.y;
    int bidx = blockIdx.x, bidy = blockIdx.y;
    int gdimx = gridDim.x;

    //int threadId = (tidy*bdimx) + tidx;
    int blockId = bidx + gdimx*bidy;
    int gthreadId = blockId*(bdimx*bdimy) + (tidy*bdimx) + tidx;

    /* finally, we have the pixel ID "pix" */
    int pix = gthreadId;
    int npix = NX * NY * L;

    if (pix >= npix)
        return;

    float x = coordx[pix];
    float y = coordy[pix];

    int tindex = 0;
    float vx[COEFFICIENT_CACHE_SIZE];
    float vy[COEFFICIENT_CACHE_SIZE];

    int ix = 0; vx[0] = 1.0f; vx[1] = x;
    for (int i=0; i<CI; i++){
        vx[ix] = vx[ix]*(ix<2) + (2.0f*vx[ix-1]*x - vx[ix-2])*(ix>=2);

        int iy = 0; vy[0] = 1.0f; vy[1] = y;
        for (int j=0; j<CJ; j++){
            vy[iy] = vy[iy]*(iy<2) + (2.0f*vy[iy-1]*y - vy[iy-2])*(iy>=2);

            tindex = i + j*CI;
            terms[tindex*npix + pix] = coef[tindex] * vx[ix] * vy[iy];

            iy++;
            // shuffle the polynomial back to the array start
            if (iy >= COEFFICIENT_CACHE_SIZE){
                vy[0] = vy[COEFFICIENT_CACHE_SIZE-2];
                vy[1] = vy[COEFFICIENT_CACHE_SIZE-1];
                iy = 2;
            }
        }

        ix++;
        // shuffle the polynomial back to the array start
        if (ix >= COEFFICIENT_CACHE_SIZE){
            vx[0] = vx[COEFFICIENT_CACHE_SIZE-2];
            vx[1] = vx[COEFFICIENT_CACHE_SIZE-1];
            ix = 2;
        }
    }
}

void init_cuda(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr, "Error: No CUDA supporting devices.\n");
        exit(EXIT_FAILURE);
    }
    hipSetDevice(CUDA_DEVICE);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, CUDA_DEVICE);

    // just to get the card warm...
    float *test;
    checkCudaErrors(hipMalloc((void**)&test, sizeof(float)));
}

static PyObject *superreg_chebpoly(PyObject *self, PyObject *args){
    PyArrayObject *coef, *coordx, *coordy;

    /* Parse parameters and verify input */
    if (!PyArg_ParseTuple(args, "O!O!O!", &PyArray_Type, &coef,
                &PyArray_Type, &coordx, &PyArray_Type, &coordy)){
        //die("Failed to parse python argument\n");
        return NULL;
    }

    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, CUDA_DEVICE);

    int CI = PyArray_DIM(coef, 0);
    int CJ = PyArray_DIM(coef, 1);
    int L = PyArray_DIM(coordx, 0);
    int NY = PyArray_DIM(coordx, 1);
    int NX = PyArray_DIM(coordx, 2);

    //printf("%i %i %i\n", L, NY, NX);
    int ncoef = CI * CJ;
    int npix = L * NX * NY;

    int Ncoef = PyArray_SIZE(coef);
    int Ncoord = PyArray_SIZE(coordx);
    uint64_t Nterms = ncoef * npix;
    int NBcoef = sizeof(float)*Ncoef;
    int NBcoord = sizeof(float)*Ncoord;
    uint64_t NBterms = sizeof(float)*Nterms;

    //printf("%i %i | %i %i | %lu\n", Ncoef, Ncoord, NBcoef, NBcoord, NBterms);

    float *h_coef, *h_coordx, *h_coordy;
    float *d_coef, *d_coordx, *d_coordy, *d_terms;
    h_coef = (float*)PyArray_DATA(coef);
    h_coordx = (float*)PyArray_DATA(coordx);
    h_coordy = (float*)PyArray_DATA(coordy);

    checkCudaErrors(hipMalloc((void**)&d_coef, NBcoef));
    checkCudaErrors(hipMalloc((void**)&d_coordx, NBcoord));
    checkCudaErrors(hipMalloc((void**)&d_coordy, NBcoord));
    checkCudaErrors(hipMalloc((void**)&d_terms, NBterms));
    checkCudaErrors(hipMemcpy(d_coef, h_coef,  NBcoef, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_coordx, h_coordx, NBcoord, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_coordy, h_coordy, NBcoord, hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());

    uint32_t WARP = 32;
    uint32_t MAXTHREADS = properties.maxThreadsPerBlock; // per dimension of block
    uint32_t MAXBLOCKS = properties.maxGridSize[0]; // per dimension of grid

    uint32_t bdimx=0, bdimy=0;
    uint32_t gdimx=0, gdimy=0;

    if (Ncoef >= MAXTHREADS){
        bdimx = MAXTHREADS;
        bdimy = 1;
    } else {
        bdimx = WARP * ceil((float)Ncoef / WARP);
        bdimy = 1;
    }

    uint32_t nblocks = ceil((float)npix / (bdimx * bdimy));

    if (nblocks > MAXBLOCKS){
        gdimx = MAXBLOCKS;
        gdimy = ceil(nblocks / MAXBLOCKS);
    } else {
        gdimx = nblocks;
        gdimy = 1;
    }

	//printf("%i %i %i %i\n", bdimx, bdimy, gdimx, gdimy);

    // actually do things
    dim3 bdim(bdimx, bdimy, 1);
    dim3 gdim(gdimx, gdimy, 1);
    checkCudaErrors(hipGetLastError());

    //printf("%i %i | %i %i\n", bdimx, bdimy, gdimx, gdimy);
    cu_chebpoly<<<gdim, bdim>>>(d_coef, d_coordx, d_coordy, d_terms, L, CI, CJ, NX, NY);
    checkCudaErrors(hipGetLastError());

    // copy data back and let it go
	float *h_terms;
	h_terms = (float*)malloc(NBterms);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_terms, d_terms, NBterms, hipMemcpyDeviceToHost));

    npy_intp dims[5] = {CJ, CI, L, NY, NX};
    PyObject *out = PyArray_SimpleNewFromData(5, dims, NPY_FLOAT32, h_terms);

    hipFree(d_coef);
    hipFree(d_coordx);
    hipFree(d_coordy);
    hipFree(d_terms);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    return out;
}








